#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2015 Matthew Nicely
 * Licensed under The MIT License (MIT)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:

 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.

 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 ******************************************************************************/

/******************************************************************************
 * wrapper.cu
 *
 * Used to call CUDA from C++ file.
 *
 ******************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "wrapper.cuh"
#include "opt_kernel.cuh"

__device__ int d_distance;

__host__ void HandleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),	file, line );
    exit( EXIT_FAILURE );
  }
} // HandleError
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

int getGPU_Info(void) {
  
  int deviceCount = 0;
  
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  
  if (error_id != hipSuccess) {
    printf( "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
  }
  
  if (deviceCount == 0) {
    printf("There is no device supporting CUDA\n");
  }
  else {
    printf("Found %d CUDA Capable device(s)\n", deviceCount);
  }
  
  int dev, driverVersion = 0, runtimeVersion = 0;     
  
  for (dev = 0; dev < deviceCount; ++dev) {
    
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    
    printf("Device %d: \"%s\"\n", dev, deviceProp.name);
    
    // Console log
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("CUDA Driver Version / Runtime Version %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
    
    printf("CUDA Capability Major/Minor version number: %d.%d\n", deviceProp.major, deviceProp.minor);
    
    printf("Total amount of global memory: %.0f MBytes (%llu bytes)\n", 
	   (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);        
    printf("Total amount of constant memory: %u bytes\n", (unsigned)deviceProp.totalConstMem); 
    printf("Total amount of shared memory per block: %u bytes\n", (unsigned)deviceProp.sharedMemPerBlock);
    printf("Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
    printf("Warp size: %d\n", deviceProp.warpSize);
    printf("Maximum number of threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
    printf("Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    printf("Maximum sizes of each dimension of a block: %d x %d x %d\n",
	   deviceProp.maxThreadsDim[0],
	   deviceProp.maxThreadsDim[1],
	   deviceProp.maxThreadsDim[2]);
    printf("Maximum sizes of each dimension of a grid: %d x %d x %d\n",
	   deviceProp.maxGridSize[0],
	   deviceProp.maxGridSize[1],
	   deviceProp.maxGridSize[2]);
    printf("\n");
    
  }
  
  return deviceCount;
}

// C++ CUDA Kernel wrapper
void cuda_function(int *route, int *distance, int num_cities, float *crap) {
  
  int blockSize;      // The launch configurator returned block size 
  int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
  int gridSize;       // The actual grid size needed, based on input size 
  
  // Create variables for GPU
  int *d_route;
  float *d_crap;
  
  // Construct and array to hold all thread-local routes.
  int *d_matrix;
  
  // Allocate memory on GPU
  HANDLE_ERROR(hipMalloc((void**)&d_route, (num_cities+1) * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&d_crap, num_cities * 3 * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&d_matrix, (num_cities-2) * (num_cities+1) * sizeof(int))); 
  
  // Memory set to zero
  HANDLE_ERROR(hipMemset(d_matrix, 0, (num_cities-2) * (num_cities+1) * sizeof(int)));
  
  // Copy from CPU to GPU
  HANDLE_ERROR(hipMemcpy(d_route, route, (num_cities+1) * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(d_crap, crap, num_cities * 3 * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_distance), distance, sizeof(int)));
  
  // Determine thread size and block size
  //   HANDLE_ERROR(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, find_route, 0, 0));
  gridSize = (num_cities + threadsPerBlock - 1) / threadsPerBlock;
  
  //   printf("minGridSize = %d\n", minGridSize);
  printf("blockSize = %d\n", threadsPerBlock);
  printf("gridSize = %d\n", gridSize);
  
  // Execute kernel
  //   find_route<<<gridSize, threadsPerBlock>>>(d_route, num_cities, d_crap, d_matrix);
  
  // Sync Device 
  HANDLE_ERROR(hipDeviceSynchronize());
  
  // Copy from GPU to CPU
  HANDLE_ERROR(hipMemcpy(route, d_route, (num_cities+1) * sizeof(int), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpyFromSymbol(distance, HIP_SYMBOL(d_distance), sizeof(int)));
  
  for (int i=0; i<num_cities+1; i++) {
    printf("%d ", route[i]);
  }
  printf("\n");
}
