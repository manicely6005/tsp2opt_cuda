/******************************************************************************
 * Copyright (c) 2015 Matthew Nicely
 * Licensed under The MIT License (MIT)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:

 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.

 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 ******************************************************************************/

/******************************************************************************
 * wrapper.cu
 *
 * Used to call CUDA from C++ file.
 *
 ******************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "wrapper.cuh"
#include "opt_kernel.cuh"

__host__ void handleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
      printf( "%s in %s at line %d\n", hipGetErrorString( err ),	file, line );
      exit( EXIT_FAILURE );
  }
} // HandleError
#define HANDLE_ERROR( err ) (handleError( err, __FILE__, __LINE__ ))

wrapper::wrapper(int num_cities)
{
  gridSize = (num_cities + threadsPerBlock - 1) / threadsPerBlock;

  //to calculate the number of jobs/2-opt changes and iteration number for each thread
  counter = (long)(num_cities-2)*(long)(num_cities-1)/2;
  iterations = (counter/(threadsPerBlock*gridSize)) + 1;

  // Allocate host memory
  h_block = new struct best_2opt[gridSize];

  // Allocate memory on GPU
  HANDLE_ERROR(hipMalloc((void**)&d_coords, num_cities * sizeof(struct city_coords)));
  HANDLE_ERROR(hipMalloc((void**)&d_block, gridSize * sizeof(struct best_2opt)));
}

wrapper::~wrapper()
{
  delete(h_block);
  // Replaces multiple hipFree's
  HANDLE_ERROR(hipDeviceReset());
}

void wrapper::getGPU_Info(void) {

  int deviceCount = 0;

  HANDLE_ERROR(hipGetDeviceCount(&deviceCount));

  if (deviceCount == 0) {
      printf("There is no device supporting CUDA\n");
  } else if (deviceCount == 1) {
      printf("Found %d CUDA Capable device\n", deviceCount);
  } else if (deviceCount > 1) {
      printf("Found %d CUDA Capable device(s)\n", deviceCount);
      printf("Setting Device to Device 1\n\n");
      HANDLE_ERROR(hipSetDevice(0));
  }
}

void wrapper::cuda_function(int num_cities, city_coords *h_coords, best_2opt *gpuResult) {

  HANDLE_ERROR(hipMemcpy(d_coords, h_coords, num_cities * sizeof(struct city_coords), hipMemcpyHostToDevice));

  // Execute kernel
  find_route<<<gridSize, threadsPerBlock>>>(num_cities, d_coords, counter, iterations, d_block);

  // Sync Device
  HANDLE_ERROR(hipDeviceSynchronize());

  // Copy from GPU to CPU
  HANDLE_ERROR(hipMemcpy(h_block, d_block, gridSize * sizeof(struct best_2opt), hipMemcpyDeviceToHost));

  // Reduction of block results
  for (int i=1; i<gridSize; i++) {
      if (h_block[i].minchange < h_block[0].minchange) h_block[0] = h_block[i];
  }

  // Copy best to structure used by two_opt()
  memcpy((void*)gpuResult, (void*)&h_block[0], sizeof(struct best_2opt));
}
