#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2015 Matthew Nicely
 * Licensed under The MIT License (MIT)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:

 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.

 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 ******************************************************************************/

/******************************************************************************
 * opt_kernel.cu
 *
 * CUDA kernel to calculate 2-opt swap on GPU.
 *
 ******************************************************************************/

#include <stdio.h>
#include "opt_kernel.cuh"
#include "wrapper.cuh"
#include "algorithms.h"

#define WIDTH 3

__global__ void find_route(int num_cities,  city_coords *coords, unsigned long long counter, unsigned int iterations) {

   __shared__ city_coords cache[MAX_CITIES];
   __shared__ int cities;
   __shared__ best_2opt best_values[threadsPerBlock];

   register int idx = threadIdx.x + blockIdx.x * blockDim.x;
   register int id;
   register unsigned int i, j;
   register unsigned long long max = counter;
   register int packSize = blockDim.x * gridDim.x;
   register unsigned int iter = iterations;
   register int change;
   register int minChange = 999999;

   cities = num_cities;

   for (register int i=threadIdx.x; i<cities; i+= blockDim.x) {
	   cache[i] = coords[i];
   }

   __syncthreads();

   // Each thread performs iter inner iterations in order to reuse the shared memory
   /* The following technique was taken from "Accelerating 2-opt and 3-opt Local Search
    * Using GPU in the Travelling Salesman Problem" by Kamil Rocki
    */

   for (register int no = 0; no < iter; no++) {
	   id = idx + no * packSize;

	   if (id < max) {
		   // Indexing Lower Triangular Matrix
		   i = (unsigned int)(3 +__fsqrt_rn(8.0f * (float) id + 1.0f))/2;
		   j = id - (i-2) * (i-1) / 2 + 1;

		   // Calculate change
//		   change =

		   // Save if local thread change is better then previous iteration best
//		   if (change < minChange) {
//			 best_value[threadIdx.x].minchange = change;
//			 best_value[threadIdx.x].i = i;
//			 best_value[threadIdx.x].j = j;
//		   }
	   }
   }

   __syncthreads();

   // Intra-block reduction
   // Reductions, threadsPerBlock must be a power of 2 because of the following code
//     register int k = blockDim.x/2;
//     while (k != 0) {
//       if (threadIdx.x < i) {
//         if (best_values[threadIdx.x + k].minchange < best_values[threadIdx.x].minchange) {
//			 	best_values[threadIdx.x].minchange = best_values[threadIdx.x + k].minchange;
//   			best_values[threadIdx.x].i = best_values[threadIdx.x + k].i;
//   			best_values[threadIdx.x].j = best_values[threadIdx.x + k].j;
//         }
//       }
//       __syncthreads();
//       i /= 2;
//     }

   __syncthreads();

   // Inter-block reduction. This will be a serial process of all blocks... Might be faster to reduce number of blocks...
   if (idx == 0) {

//	   if (best_values[threadIdx.x].minchange < best.minchange) {
//		   // Atomic function
//		   atomicMin(&(best.minchange), best_values[threadIdx.x].minchange)
//	   }

	   printf("best = %d, %d, %d\n", best.i, best.j, best.minchange);\
	   best.i = best.i + 5;
	   best.minchange  = 1000;

	   for (int i=0; i<cities; i++) {
		   printf("cache[i].x = %f ", cache[i].x);
	       printf("cache[i].y = %f\n", cache[i].y);
	   }
   }
}

/* This is a wrapper function which allows the wrapper file to copy to a symbol
 * This is because hipMemcpyToSymbol is implicit local scope linkage. Meaning
 * hipMemcpyToSymbol must be in the same generated .obj file of the kernel
 * where you want to use it. Link to more info below.
 * http://stackoverflow.com/questions/16997611/cuda-writing-to-constant-memory-wrong-value */
__host__ void setParam(struct best_2opt zero) {
	hipMemcpyToSymbol(HIP_SYMBOL(best), &zero, sizeof(struct best_2opt));
}

/* This is a wrapper function which allows the wrapper file to copy to a symbol
 * This is because hipMemcpyToSymbol is implicit local scope linkage. Meaning
 * hipMemcpyToSymbol must be in the same generated .obj file of the kernel
 * where you want to use it. Link to more info below.
 * http://stackoverflow.com/questions/16997611/cuda-writing-to-constant-memory-wrong-value */
__host__ void getParam(struct best_2opt * out) {
	hipMemcpyFromSymbol(out, HIP_SYMBOL(best), sizeof(struct best_2opt));
}

// __device__ void geo(int idx, int *matrix, int num_cities, float *crap, int *distance) {
//   
//   int deg, j;
//   double xi, yi, xj, yj;
//   double PI = 3.141492;
//   double min, latitude_i, latitude_j, longitude_i, longitude_j, RRR, q1, q2, q3;
//   
//   for (int i=0; i<num_cities; i++) {
//     j = i + 1;
//     
//     // matrix[i] - 1 convert the 1 based matrix to the 0 based crap
//     xi = crap[(matrix[idx*(num_cities+1)+i] - 1)*WIDTH+1];    // x coordinate
//     yi = crap[(matrix[idx*(num_cities+1)+i] - 1)*WIDTH+2];    // y coordinate
//     xj = crap[(matrix[idx*(num_cities+1)+j] - 1)*WIDTH+1];    // x coordinate
//     yj = crap[(matrix[idx*(num_cities+1)+j] - 1)*WIDTH+2];    // y coordinate
//     
//     //     printf("xi = %f : yi = %f : xj = %f : yj = %f\n", xi, yi, xj, yj);
//     
//     deg = (int) xi;
//     min = xi - deg;
//     latitude_i = PI * (deg + 5.0 * min/3.0)                                                                 / 180.0;
//     
//     deg = (int) yi;
//     min = yi - deg;
//     longitude_i = PI * (deg + 5.0 * min/3.0) / 180.0;
//     
//     deg = (int) xj;
//     min = xj - deg;
//     latitude_j = PI * (deg + 5.0 * min/3.0) / 180.0;
//     
//     deg = (int) yj;
//     min = yj - deg;
//     longitude_j = PI * (deg + 5.0 * min/3.0) / 180.0;
//     
//     // The distance between two different nodes i and j in kilometers is then computed as follows:
//     RRR = 6378.388;
//     
//     q1 = cos(longitude_i - longitude_j);
//     q2 = cos(latitude_i - latitude_j);
//     q3 = cos(latitude_i + latitude_j);
//     
//     *distance += (int) (RRR * acos(0.5 * ((1.0 + q1) * q2 - (1.0 - q1) * q3)) + 1.0);
//   }
// }
